#include "hip/hip_runtime.h"
#include <optix_world.h> // Include access to the C++ API in the optix namespaces.

using namespace optix;

// Configuration variables
rtDeclareVariable(float3, origin, , );// The origin of this vertical cylinder (in the x,z plane)
rtDeclareVariable(float, radius, , );// The radius of this vertical cylinder
rtDeclareVariable(float, height, , );
rtDeclareVariable(float2, textureCoords, attribute textureCoords, );
rtDeclareVariable(float, cameraPositionFromGround, , );

// Precalculated variables

// Required variables
rtDeclareVariable(int, lgt_instance, , ) = {0};
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(int, lgt_idx, attribute lgt_idx, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

static __device__ float2 to2D(float3 v)
{
  return(make_float2(v.x, v.z));
}
RT_PROGRAM void intersect(int)
{
  //// Calculate the intersection of the ray with the circle in 2D:

  float2 rayOrigin2D = to2D(ray.origin);//make_float2(ray.origin.x, ray.origin.z);
  float2 rayDir2D = to2D(ray.direction);//make_float2(ray.direction.x, ray.direction.z);
  float2 origin2D = to2D(origin);
  float2 cylinderToRay = origin2D - rayOrigin2D;
  float a = dot(rayDir2D, rayDir2D);
  if(a == 0.0f)
    return; // The ray does not intersect the cylinder.
  else
    a *= 2;// Multiply a by two to turn it into the denominator
  float b = -2*dot(rayDir2D, cylinderToRay); // Multiply by -1 so that it doesn't need to be later (disc uses b^2)
  float c = dot(cylinderToRay, cylinderToRay) - radius * radius;

  float disc = (b*b)-(2*a*c);

  //if(threadIdx.x == 10)
  //  printf("%f\n", disc);

  if(disc < 0 || disc == 0.0f)
    return; // The ray does not intersect the cylinder or the ray intersects but only at the edges, which we don't care about.

  //if(threadIdx.x == 10)
  //  printf("2\n");

  // The ray intersects the cylinder two times.
  // Now we can caluclate the two points on the circle. As we only care about rays coming
  // from the inside out, we can calculate each variation of the root and take the first
  // one that dots positively with the surface normal, meaning that it's viewing from
  // the inside.
  
  // Derive the correct non-backface intersect point:
  float sqrtDisc = sqrt(disc);
  float distance = (-b + sqrtDisc)/a;
  float2 intersectPoint2D = rayOrigin2D + distance * rayDir2D;
  float2 normal2D = normalize(origin2D - intersectPoint2D);
  float cullTest = dot((rayOrigin2D - intersectPoint2D), normal2D);
  if(cullTest < 0)// If the point is a back-face, then we'll have to calculate using the other point...
  {
    distance = (-b - sqrtDisc)/a;
    intersectPoint2D = rayOrigin2D + distance * rayDir2D;
    normal2D = normalize(origin2D - intersectPoint2D);
  }
  
  // Convert the 2D coordinates to 3D ones:
  float3 intersectPoint = ray.origin + distance * ray.direction;
  float3 normal = make_float3(normal2D.x, 0.0f, normal2D.y);

  // Render the object if the intersect point is within the accepted vertical space:
  if(intersectPoint.y >= origin.y && intersectPoint.y <= origin.y + height && rtPotentialIntersection(distance))
  {
    shading_normal = geometric_normal = normal;
    // Calculate the texture coordinates
    textureCoords = make_float2((atan2f(intersectPoint2D.x-origin2D.x, intersectPoint2D.y-origin2D.y)+M_PIf)/M_PIf/2.0f, (intersectPoint.y - origin.y)/height);
    lgt_idx = lgt_instance;
    rtReportIntersection(0);// Render.
  }
}

RT_PROGRAM void bounds(int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  printf("Hello from verticalCylinder Bounds\n");
  float3 maxArea = origin + make_float3(radius, height, radius);
  float3 minArea = origin - make_float3(radius, 0.0f, radius);
  printf("  | maxArea: (%f, %f, %f)\n", maxArea.x, maxArea.y, maxArea.z);
  printf("  | minArea: (%f, %f, %f)\n", minArea.x, minArea.y, minArea.z);
  aabb->set(minArea, maxArea);
}
