#include "hip/hip_runtime.h"
#include <optix_world.h> // Include access to the C++ API in the optix namespace

using namespace optix;

// Configuration variables
rtDeclareVariable(float3, origin, , ); // The origin of this sphere.
rtDeclareVariable(float, radius, , ); // The radius of this sphere.

// Precalculated variables

// Recuired variables
rtDeclareVariable(int, lgt_instance, , ) = {0};
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(int, lgt_idx, attribute lgt_idx, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int)
{
  //if(threadIdx.x == 10)
  //  printf("Hello from the sphere.\n");
  float3 rayToOrigin = ray.origin - origin;// - ray.origin;
  float a = dot(ray.direction, ray.direction);
  if(a == 0)
    return; // Return if a would induce a div 0.
  else
    a *= 2;// Pre-multiply a by 2 to avoid potential double-calculation later.
  float b = -2 * dot(ray.direction, rayToOrigin);// Pre-multiply b by -1 to avoid potential double-calculation later.
  float c = dot(rayToOrigin, rayToOrigin) - radius*radius;

  // Calculate intersection distances:
  float discriminator = b*b - 2*a*c;
  if(discriminator <= 0)
    return; // Return if the ray doesn't intersect (or it only grazes the outside, it's not worth calculating that for this.

  // Calculate the first intersection
  discriminator = sqrt(discriminator);// Perform sqrt.
  float distance;
  if(discriminator < 0)// If the disc. is < 0, then the closer variant (smaller distance) is found from adding the srt to b, then dividing.
    distance = (b + discriminator)/a;
  else
    distance = (b - discriminator)/a;// If not, then subtract.
  float3 intersectPoint = ray.origin + distance * ray.direction;

  if(rtPotentialIntersection(distance))
  {
    shading_normal = geometric_normal = normalize(intersectPoint - origin);// The normal radiates from the centre of the sphere.
    lgt_idx = lgt_instance;
    rtReportIntersection(0);// Render
  }
}
RT_PROGRAM void bounds(int, float result[6])
{
  //printf("Hello from sphere bounds\n");
  optix::Aabb* aabb = (optix::Aabb*)result;
  float3 maxPoint = origin + make_float3(radius);
  float3 minPoint = origin - make_float3(radius);
  //printf("  | maxPoint: (%f, %f, %f)\n", maxPoint.x, maxPoint.y, maxPoint.z);
  //printf("  | minPoint: (%f, %f, %f)\n", minPoint.x, minPoint.y, minPoint.z);
  aabb->set(minPoint, maxPoint);
}
