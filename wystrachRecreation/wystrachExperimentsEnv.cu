#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "tutorial.h"

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(float2, uv_coords,        attribute uv_coords, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );


///// CAMERA SHADERS

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float2,        viewRange, , );// The range of viewing for the 360-degree eye, (0-360, 0-180)
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float3,        renderPosition, , );// The position of the 360-degree-experimental camera.
rtBuffer<uchar4, 2>              output_buffer;

const float ONEHUNDREDTWENTY_DEGREES = 120.0f/180.0f * M_PIf;
const float THIRTY_DEGREES = 30.0f/180.0f * M_PIf;

// A camera to perform the 360 degree FOV renders.
rtDeclareVariable(int, legal, , );
RT_PROGRAM void threeSixtyExperimentCamera()
{
  if(!legal)
  {
    output_buffer[launch_index] = make_color(bad_color);
    return;
  }
  size_t2 screen = output_buffer.size();
  // Full panorama:
  //float2 d = make_float2(launch_index) / make_float2(screen) * make_float2(2.0f * M_PIf, M_PIf) - make_float2(M_PIf, M_PIf/2.0f);
  // 30 degree from the floor (vertical range of -30 deg to 90 deg):
  float2 d = make_float2(launch_index) / make_float2(screen) * make_float2(2.0f * M_PIf, ONEHUNDREDTWENTY_DEGREES) - make_float2(M_PIf, THIRTY_DEGREES);
  float3 ray_origin = renderPosition;
  float3 ray_direction = make_float3(sin(d.x)*cos(d.y), sin(d.y), cos(d.x)*cos(d.y));// This should already be normalized, as it is a sphere.

  optix::Ray ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon);

  PerRayData_radiance prd;
  prd.importance = 1.0f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  output_buffer[launch_index] = make_color(prd.result);
}
RT_PROGRAM void threeSixty_camera()
{
  size_t2 screen = output_buffer.size();

  //// ORIGINAL:
  //float2 d = make_float2(launch_index) / make_float2(screen) * make_float2(2.0f * M_PIf , M_PIf) + make_float2(M_PIf, 0);

  float2 d = make_float2(launch_index) / make_float2(screen) * make_float2(1.f * M_PIf , M_PIf) + make_float2(M_PIf, 0)
                                                             + make_float2(1.f * M_PIf, 0);
  float3 angle = make_float3(cos(d.x) * sin(d.y), -cos(d.y), sin(d.x) * sin(d.y));
  float3 ray_origin = eye;
  float3 ray_direction = normalize(angle.x*normalize(U) + angle.y*normalize(V) + angle.z*normalize(W));

  ////float2 d = make_float2(launch_index)/make_float2(screen) * viewRange + (viewRange * make_float2(0.75f, 0.f));
  //float2 d = make_float2(launch_index)/make_float2(screen) * make_float2(2.f, 2.f) - make_float2(screen);//viewRange + (viewRange * make_float2(0.75f, 0.f));
  ////if(viewRange == 0.0f)
  ////{
  ////  d = d * make_float2(2.0f * M_PIf , M_PIf) + make_float2(M_PIf, 0.f);// Default 360 degree configuration
  ////}else{
  ////  d = d * make_float2(
  ////}
  ////float2 d = make_float2(launch_index) / make_float2(screen) * make_float2(2.0f * M_PIf , M_PIf/2.0f) + make_float2(M_PIf, M_PIf/4.0f);
  //float3 angle = make_float3(cos(d.x) * sin(d.y), -cos(d.y), sin(d.x) * sin(d.y));
  //float3 ray_origin = eye;
  //float3 ray_direction = normalize(angle.x*normalize(U) + angle.y*normalize(V) + angle.z*normalize(W));


  //if(threadIdx.x == 10)
  //{
  //  printf("Launch index: (%f, %f), screen: (%f, %f), d: (%f, %f)\n", lidx.x, lidx.y, scrn.x, scrn.y, d.x, d.y);
  //}
  // Apply camera rotation using U, V & W rotation information.
  //float3 ray_direction = normalize(d.x*make_float3(1,0,0) + d.y*make_float3(0,1,0) + make_float3(0,0,1)); //normalize(d.x*U + d.y*V + W);

  optix::Ray ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon );

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  output_buffer[launch_index] = make_color( prd.result );
}


RT_PROGRAM void pinhole_camera()
{
  size_t2 screen = output_buffer.size();

  float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);

  optix::Ray ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon );

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  output_buffer[launch_index] = make_color( prd.result );
}

RT_PROGRAM void deformed_camera()
{
  size_t2 screen = output_buffer.size();

  float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
  float3 offset = normalize(make_float3(d.x,d.y, 0.f));
  float3 ray_origin = eye + 2.f*normalize(offset.x*U + offset.y*V + W);
  //float3 ray_origin = normalize(offset.x*U + offset.y*V + W);
  //float3 ray_origin = eye + normalize(make_float3(d.x,d.y, 0.f));
  float3 ray_direction = normalize(d.x*U + d.y*V + W);

  optix::Ray ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon );

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  output_buffer[launch_index] = make_color( prd.result );
}


///// MISS SHADERS

rtDeclareVariable(float3, bg_color, , );
// The default miss program
RT_PROGRAM void miss()
{
  prd_radiance.result = bg_color;
}
// Utilises an ENV map
rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void missEnv()
{
  float theta = atan2f(ray.direction.x, ray.direction.z);
  float phi = M_PIf * 0.5f - acosf(ray.direction.y);
  float u = (theta + M_PIf) * (0.5f * M_1_PIf);
  float v = 0.5f * (1.0f + sin(phi));
  prd_radiance.result = make_float3(tex2D(envmap, u, v));
}
// First Ofstad miss attempt
RT_PROGRAM void missOfstad()
{
  float theta = atan2f(ray.direction.x, ray.direction.z);
  float phi = M_PIf * 0.5f - acosf(ray.direction.y);
  float u = (theta + M_PIf) * (0.5f * M_1_PIf);
  float v = 0.5f * (1.0f + sin(phi));
  //prd_radiance.result = make_float3(tex2D(envmap, u, v));
  //prd_radiance.result = bg_color;
  if(theta < M_PIf/4.f)
  {
    if ((int)(theta*3)%2 == 0)
    {
      prd_radiance.result = make_float3(0,0,0);
    }else{
      prd_radiance.result = make_float3(1,1,1);
    }
  }else{
    if ((int)(phi*4)%2 == 0)
    {
      prd_radiance.result = make_float3(0,0,0);
    }else{
      prd_radiance.result = make_float3(1,1,1);
    }
  }
}
  

//
// Terminates and fully attenuates ray after any hit
//
RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = make_float3(0);

  rtTerminateRay();
}
  

///// SURFACE SHADERS

rtDeclareVariable(float3,   ambient_light_color, , ); // Stores the ambient light colour of anyting
RT_PROGRAM void solid_color()
{
  prd_radiance.result = ambient_light_color;
}

rtBuffer<BasicLight>        lights;
rtDeclareVariable(float, base_light_mult, , );// Sometimes called K_a
rtDeclareVariable(float3,   Kd, , );
RT_PROGRAM void shaded_solid_color()
{
  float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal));
  float3 hit_point = ray.origin + t_hit * ray.direction;
  float3 color = base_light_mult * ambient_light_color;

  // For each light, calculate it's affect on the ambient light color.
  for(int i = 0; i < lights.size(); i++)
  {
    BasicLight light = lights[i];
    float3 dirToLight = normalize(light.pos - hit_point);
    float lightScale = dot(dirToLight, world_shade_normal); // (a.b)/(|a||b|) = cos(@), but here |a| and |b| have been made to equal 1.
    color += light.color * lightScale * Kd;
  }
  prd_radiance.result = color;
}

rtDeclareVariable(float2, textureCoords, attribute textureCoords , );
const float ONE_THIRD = 1.0f/3.0f;
const float TWO_THIRDS = 2.0f/3.0f;
RT_PROGRAM void ofstad_cylinder()
{
  //printf("texture: (%f, %f)\n", textureCoords.x, textureCoords.y);
  float3 colour = make_float3(textureCoords.x,textureCoords.y,0.0f);// Default colour displays coordinates.
  if(textureCoords.x <= ONE_THIRD)
  {
    // Vertical Bars
    colour = make_float3((int)floor(textureCoords.x/ONE_THIRD*8)%2 == 0);
  }else if(textureCoords.x <= TWO_THIRDS){
    // Horizontal Bars
    colour = make_float3((int)floor(textureCoords.y/ONE_THIRD)%2 == 0);
  }else{
    // Diagonal Bars
    colour = make_float3((int)floor(textureCoords.y/ONE_THIRD+ (textureCoords.x-TWO_THIRDS)/ONE_THIRD*8)%2 == 1);
  }
  prd_radiance.result = colour;
}

rtDeclareVariable(float3, virtualCameraPosition, , );
rtDeclareVariable(int, tint, , );
RT_PROGRAM void dewarNaturalPanorama()
{
  PerRayData_radiance returned_prd;
  returned_prd.importance = 1.0f;
  returned_prd.depth = prd_radiance.depth+1;

  float3 hitPoint = ray.origin + t_hit * ray.direction;
  //float3 spreadRayDir = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, -geometric_normal) );
  float3 spreadRayDir = normalize(hitPoint - virtualCameraPosition);
  hitPoint = hitPoint + 0.001f * spreadRayDir;
  optix::Ray spreadRay(hitPoint, spreadRayDir, RADIANCE_RAY_TYPE, scene_epsilon);
  rtTrace(top_object, spreadRay, returned_prd);
  
  float3 colour = returned_prd.result;
  if(tint == 1)
    colour += make_float3(0.0f,0.1f,0.0f);

  prd_radiance.result = colour;
}

rtTextureSampler<float4, 2> treemap;
RT_PROGRAM void treeBillboard()
{
  float3 textureColour = make_float3(tex2D(treemap, uv_coords.x, uv_coords.y));

  PerRayData_radiance returned_prd;
  returned_prd.importance = 1.0f;
  returned_prd.depth = prd_radiance.depth+1;
  float3 hitPoint = ray.origin + t_hit * ray.direction;
  optix::Ray continuedRay(hitPoint, ray.direction, RADIANCE_RAY_TYPE, scene_epsilon);
  rtTrace(top_object, continuedRay, returned_prd);
  textureColour = returned_prd.result * textureColour;

  prd_radiance.result = textureColour;
}


//
// Phong surface shading with shadows 
//
rtDeclareVariable(float3,   Ka, , );
rtDeclareVariable(float3,   Ks, , );
rtDeclareVariable(float,    phong_exp, , );
rtDeclareVariable(rtObject, top_shadower, , );

RT_PROGRAM void closest_hit_radiance3()
{
  float3 world_geo_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal     = faceforward( world_shade_normal, -ray.direction, world_geo_normal );
  float3 color = Ka * ambient_light_color;

  float3 hit_point = ray.origin + t_hit * ray.direction;

  for(int i = 0; i < lights.size(); ++i) {
    BasicLight light = lights[i];
    float3 L = normalize(light.pos - hit_point);
    float nDl = dot( ffnormal, L);

    if( nDl > 0.0f ){
      // cast shadow ray
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = make_float3(1.0f);
      float Ldist = length(light.pos - hit_point);
      optix::Ray shadow_ray( hit_point, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist );
      rtTrace(top_shadower, shadow_ray, shadow_prd);
      float3 light_attenuation = shadow_prd.attenuation;

      if( fmaxf(light_attenuation) > 0.0f ){
        float3 Lc = light.color * light_attenuation;
        color += Kd * nDl * Lc;

        float3 H = normalize(L - ray.direction);
        float nDh = dot( ffnormal, H );
        if(nDh > 0)
          color += Ks * Lc * pow(nDh, phong_exp);
      }

    }
  }
  prd_radiance.result = color;
}
  

//
// (NEW)
// Phong surface shading with shadows and reflections
//
rtDeclareVariable(float3, reflectivity, , );
rtDeclareVariable(float, importance_cutoff, , );
rtDeclareVariable(int, max_depth, , );

RT_PROGRAM void floor_closest_hit_radiance4()
{
  float3 world_geo_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal     = faceforward( world_shade_normal, -ray.direction, world_geo_normal );
  float3 color = Ka * ambient_light_color;

  float3 hit_point = ray.origin + t_hit * ray.direction;

  float3 wobble;
  float mult = 1.0f;
  wobble.x = sin(hit_point.x * mult);
  wobble.y = cos(hit_point.z * mult);
  wobble.z = sin(hit_point.z * mult);
  world_shade_normal = world_shade_normal + wobble;
  world_shade_normal = world_geo_normal + wobble;

  float3 true_point;
  true_point.x = hit_point.x;
  //true_point.y = hit_point.y + ((cos(hit_point.x) * sin(hit_point.z) + 1)/2 );
  true_point.y = hit_point.y + (wobble.x * wobble.y + mult * mult)/(mult/2);
  true_point.z = hit_point.z;

  for(int i = 0; i < lights.size(); ++i) {
    BasicLight light = lights[i];
    float3 L = normalize(light.pos - true_point);
    float nDl = dot( ffnormal, L);

    if( nDl > 0.0f ){
      // cast shadow ray
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = make_float3(1.0f);
      float Ldist = length(light.pos - true_point);
      optix::Ray shadow_ray( true_point, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist );
      rtTrace(top_shadower, shadow_ray, shadow_prd);
      float3 light_attenuation = shadow_prd.attenuation;

      if( fmaxf(light_attenuation) > 0.0f ){
        float3 Lc = light.color * light_attenuation;
        color += Kd * nDl * Lc;

        float3 H = normalize(L - ray.direction);
        float nDh = dot( ffnormal, H );
        if(nDh > 0)
          color += Ks * Lc * pow(nDh, phong_exp);
      }

    }
  }

  float importance = prd_radiance.importance * optix::luminance( reflectivity );


  //wobble = normalize(wobble);
  //wobble.x = (wobble.x + 1)/2;
  //wobble.y = (wobble.y + 1)/2;
  //wobble.z = (wobble.z + 1)/2;
  // reflection ray
  if( importance > importance_cutoff && prd_radiance.depth < max_depth) {
    PerRayData_radiance refl_prd;
    refl_prd.importance = importance;
    refl_prd.depth = prd_radiance.depth+1;
    float3 R = reflect( ray.direction, ffnormal );
    /// Zeroed vector
    //float3 zeroed;
    //zeroed.x = 1.0f;
    //zeroed.y = 0.0f;
    //zeroed.z = 1.0f;
    //R = R * zeroed;
    /// All reflect toward the box
    //float3 boxPos;
    //boxPos.x = 0.0f;
    //boxPos.y = 3.5f;
    //boxPos.z = 0.0f;
    //float3 R = normalize(boxPos-hit_point);
    /// Sinusoidal surface wobble
    R += wobble * 0.1;
    optix::Ray refl_ray( hit_point, R, RADIANCE_RAY_TYPE, scene_epsilon );
    rtTrace(top_object, refl_ray, refl_prd);
    //color += reflectivity * refl_prd.result;
    color = refl_prd.result; // Full reflectivity
  }

  prd_radiance.result = color;
  //prd_radiance.result = wobble;
}
  

//
// Set pixel to solid color upon failure
//
RT_PROGRAM void exception()
{
  output_buffer[launch_index] = make_color( bad_color );
}
