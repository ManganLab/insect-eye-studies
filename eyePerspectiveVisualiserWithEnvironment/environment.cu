#include "hip/hip_runtime.h"
#include "tutorial.h"

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(float2, uv_coords,        attribute uv_coords, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );

///// OMMATIDIAL POSITION STORAGE
rtBuffer<float3> ommatidia;


///// CAMERA SHADERS

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtBuffer<uchar4, 2>              output_buffer;
rtDeclareVariable(float3,        renderPosition, , );// The position of the ommatidial camera.

const float ONEHUNDREDTWENTY_DEGREES = 120.0f/180.0f * M_PIf;
const float THIRTY_DEGREES = 30.0f/180.0f * M_PIf;


// A simple pinhole camera
RT_PROGRAM void pinhole_camera()
{
  size_t2 screen = output_buffer.size();

  float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);

  optix::Ray ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon );

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  output_buffer[launch_index] = make_color( prd.result );
}

// A camera that only outputs green
RT_PROGRAM void green_camera()
{
  output_buffer[launch_index] = make_color(make_float3(0.0f,1.0f,0.0f));
}

//float radialDistancce = 
// A camera that finds the closest ommatidium to each point as a point on a sphere and renders from it's perspective
RT_PROGRAM void ommatidial_camera()
{
  size_t2 screen = output_buffer.size();
  // Full panorama:
  float2 d = make_float2(launch_index) / make_float2(screen) * make_float2(2.0f * M_PIf, M_PIf) - make_float2(M_PIf, M_PIf/2.0f);
  // 30 degree from the floor (vertical range of -30 deg to 90 deg):
  //float2 d = make_float2(launch_index) / make_float2(screen) * make_float2(2.0f * M_PIf, ONEHUNDREDTWENTY_DEGREES) - make_float2(M_PIf, THIRTY_DEGREES);
  float3 ray_direction = make_float3(sin(d.x)*cos(d.y), sin(d.y), cos(d.x)*cos(d.y));// This should already be normalized, as it is a sphere.

  // ray_direction now acts as a point on the sphere to render to, the closest ommatidium must be rendered now (TODO: in the future, pre-render each and do this bit in a separate shader)

  float closestDistance = acos(dot(ommatidia[0], ray_direction));
  int closestIndex = 0;
  float dist;
  int i;
  for(i = 1; i<ommatidia.size(); i++)
  {
    dist = acos(dot(ommatidia[i], ray_direction));
    if(dist < closestDistance)
    {
      closestDistance = dist;
      closestIndex = i;
    }
  }

  /// DEBUG
  //if(threadIdx.x == 10)
  //{
  //  //printf("Launch index: (%f, %f), screen: (%f, %f), d: (%f, %f)\n", lidx.x, lidx.y, scrn.x, scrn.y, d.x, d.y);
  //  printf("RAY LISTING....\n");
  //  printf("%f\n", closestDistance);
  //  for(i = 1; i<ommatidia.size(); i++)
  //  {
  //    printf("[%i] (%.2f, %.2f, %.2f)\n", i, ommatidia[i].x, ommatidia[i].y, ommatidia[i].z);
  //  }
  //}
#ifdef DEBUG
  if(closestDistance < 0.01f)
  {
    output_buffer[launch_index] = make_color(make_float3(1.0f,0.0f,0.0f));
  }else{
    optix::Ray ray(renderPosition, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon);

    PerRayData_radiance prd;
    prd.importance = 1.0f;
    prd.depth = 0;

    rtTrace(top_object, ray, prd);

    output_buffer[launch_index] = make_color(prd.result);
  }
#else
  optix::Ray ray(renderPosition, ommatidia[closestIndex], RADIANCE_RAY_TYPE, scene_epsilon);
  PerRayData_radiance prd;
  prd.importance = 1.0f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  output_buffer[launch_index] = make_color(prd.result);
#endif
}

///// MISS SHADERS

rtDeclareVariable(float3, bg_color, , );
// The default miss program
RT_PROGRAM void miss()
{
  prd_radiance.result = bg_color;
}

rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void miss_env()
{
  float theta = atan2f(ray.direction.x, ray.direction.z);
  float phi = M_PIf * 0.5f - acosf(ray.direction.y);
  float u = (theta + M_PIf) * (0.5f * M_1_PIf);
  float v = 0.5f * (1.0f + sin(phi));
  prd_radiance.result = make_float3(tex2D(envmap, u, v));
  //prd_radiance.result = make_float3(1.0f, 0.0f, 0.0f);
}

//
// Terminates and fully attenuates ray after any hit
//
RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = make_float3(0);

  rtTerminateRay();
}
  

///// SURFACE SHADERS


////// THERE CAN ONLY BE ONNNNE!!!
rtDeclareVariable(float3,   ambient_light_color, , ); // Stores the ambient light colour of anyting
RT_PROGRAM void solid_color()
{
  prd_radiance.result = ambient_light_color;
}

//
// Set pixel to solid color upon failure
//
RT_PROGRAM void exception()
{
  output_buffer[launch_index] = make_color( bad_color );
}
