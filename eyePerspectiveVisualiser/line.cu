#include "hip/hip_runtime.h"
#include <optix_world.h> // Include access to the C++ API in the optix namespace

using namespace optix;

// Configuration variables
rtDeclareVariable(float3, origin, , ); // The origin of this line
rtDeclareVariable(float3, direction, , ); // The direction of this line
rtDeclareVariable(float, lineLength, , ); // The length of this line
rtDeclareVariable(float, radius, , ); // The thickness of this line

// Recuired variables
rtDeclareVariable(int, lgt_instance, , ) = {0};
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(int, lgt_idx, attribute lgt_idx, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int)
{
  //The minimum distance between two lines in 3D is equal to the distance between two planes containing those lines
  float3 normal = cross(ray.direction, direction); // Form the plane normals
  float distance = dot(ray.origin - origin, normal)/dot(normal, normal);// Distance between the two planes.

  if(distance > radius)
    return; // This ray missed.

  /// Now we know the ray gets within the tube of this line, we need to work out where the collision is.
  /// Project the ray direction and origin down to a 2D plane formed with the line direction as the normal
  // Generate x and y axis of 2D line-space.
  float3 xAxis = normalize(cross(direction, make_float3(0.0f,1.0f,0.0f)));
  float3 yAxis = normalize(cross(xAxis, direction));
  // Project the direction and origin (as in, the origin from the new coordinate system)
  float2 projectedRayDirection = make_float2(dot(ray.direction, xAxis), dot(ray.direction, yAxis));
  float3 relativeOrigin = ray.origin - origin;
  float2 projectedRelativeRayOrigin = make_float2(dot(relativeOrigin, xAxis), dot(relativeOrigin, yAxis));
  /// calculate the distance from the ray origin to the hit point in 2D (where the ray is exactly 'radius' away from the line)
  // The length of the projected relative ray origin^2 - d^2 = length to hit point in 2D space
  // Note that here we're not using length() because that would return a sqrt'd value that we'd have to square again, so we calculate the squared length of projectedRelativeRayOrigin manually.
  float distanceTo2Dhit = sqrt((projectedRelativeRayOrigin.x*projectedRelativeRayOrigin.x + projectedRelativeRayOrigin.y*projectedRelativeRayOrigin.y) - radius*radius);
  /// calculate the scale factor (distance to the 3D hit) for the projected ray origin to get it to the hit point, this is the distance.
  float distanceTo3Dhit = distanceTo2Dhit/length(projectedRayDirection);

  //if(rtPotentialIntersection(distanceTo3Dhit))
  if(rtPotentialIntersection(0.001f))
  {
    shading_normal = geometric_normal = make_float3(0.0f, 1.0f, 0.0f); // For now.
    lgt_idx = lgt_instance;
    rtReportIntersection(0);// Render.
  }
}
RT_PROGRAM void bounds(int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  float3 end = origin + direction * lineLength;
  float3 tipRadius = make_float3(radius);
  aabb->set(fminf(origin, end) - tipRadius, fmaxf(origin, end) + tipRadius);
}
