#include "hip/hip_runtime.h"
#include "tutorial.h"

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(float2, uv_coords,        attribute uv_coords, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );


///// CAMERA SHADERS

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtBuffer<uchar4, 2>              output_buffer;


// A simple pinhole camera
RT_PROGRAM void pinhole_camera()
{
  size_t2 screen = output_buffer.size();

  float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);

  optix::Ray ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon );

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  output_buffer[launch_index] = make_color( prd.result );
}

///// MISS SHADERS

rtDeclareVariable(float3, bg_color, , );
// The default miss program
RT_PROGRAM void miss()
{
  prd_radiance.result = bg_color;
}

//
// Terminates and fully attenuates ray after any hit
//
RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = make_float3(0);

  rtTerminateRay();
}
  

///// SURFACE SHADERS


////// THERE CAN ONLY BE ONNNNE!!!
rtDeclareVariable(float3,   ambient_light_color, , ); // Stores the ambient light colour of anyting
RT_PROGRAM void solid_color()
{
  prd_radiance.result = ambient_light_color;
}

//
// Set pixel to solid color upon failure
//
RT_PROGRAM void exception()
{
  output_buffer[launch_index] = make_color( bad_color );
}
