#include "hip/hip_runtime.h"
#include "tutorial.h"

//// Required Variables
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(float2, uv_coords,        attribute uv_coords, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );

rtDeclareVariable(float3, base_color, , ); // Stores the base colour of this object
rtDeclareVariable(float3, ambient_illumination, , ); // Stores the ambient colour of the environment
rtDeclareVariable(float3, sun_color, , );// The colour of the sun above the object
rtDeclareVariable(float3, sun_direction, , );// The direction of the sun from the object
RT_PROGRAM void basic_shaded_solid_color()
{
  float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal));
  float3 color = base_color * ambient_illumination;

  color += sun_color * base_color * fmaxf(0.0f, dot(sun_direction, world_shade_normal));
  prd_radiance.result = color;
}

rtTextureSampler<float4, 2> Kd_map;
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 

RT_PROGRAM void basic_shaded_texture()
{
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  
  float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

  const float3 Kd_val = make_float3( tex2D( Kd_map, texcoord.x, texcoord.y ) );

  // At this point Kd_val is treated as the base color, ffnormal is the world_shade_normal (it's been modified by the vertex data)
  float3 color = Kd_val * ambient_illumination;
  color += sun_color * Kd_val * fmaxf(0.0f, dot(sun_direction, ffnormal));
  prd_radiance.result = color;
}
