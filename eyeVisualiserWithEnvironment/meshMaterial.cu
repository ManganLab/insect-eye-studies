#include "hip/hip_runtime.h"
#include "tutorial.h"

//// Required Variables
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(float2, uv_coords,        attribute uv_coords, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );

rtDeclareVariable(float3, base_color, , ); // Stores the base colour of this object
rtDeclareVariable(float3, ambient_illumination, , ); // Stores the ambient colour of the environment
rtDeclareVariable(float3, sun_color, , );// The colour of the sun above the object
rtDeclareVariable(float3, sun_direction, , );// The direction of the sun from the object
RT_PROGRAM void basic_shaded_solid_color()
{
  float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal));
  float3 color = base_color * ambient_illumination;
  float3 lightDirection = make_float3(0.0f, 1.0f, 0.0f); // The light is directly above

  color += sun_color * base_color * fmaxf(0.0f, dot(lightDirection, world_shade_normal));
  prd_radiance.result = color;
}


